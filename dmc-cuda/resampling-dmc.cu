#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <moderngpu/kernel_mergesort.hxx>
#include <moderngpu/kernel_sortedsearch.hxx>
#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_reduce.hxx>
#include <moderngpu/kernel_scan.hxx>

#include "random-numbers.hxx"

int main(int argc, char **argv) {
  mgpu::standard_context_t context;
  uint seed = 11;
  uint num_walkers = 20;

  mgpu::mem_t<float> weights(num_walkers, context);
  mgpu::mem_t<float> total_weight(1, context);
  
  auto weights_data = weights.data();
  mgpu::transform_reduce(
    [=]MGPU_DEVICE(uint index) {
      //float random = gpu_random::uniforms<1>(uint4{index, 0, 0, 0}, uint2{seed, 0})[0];
      float random = index*index;
      //random *= random;
      weights_data[index] = random;
      return random;
    },
    num_walkers,
    total_weight.data(),
    mgpu::plus_t<float>(),
    context);

  float total_weight_host = mgpu::from_mem(total_weight)[0];
  printf("%f\n", total_weight_host);

  mgpu::transform(
    [=]MGPU_DEVICE(uint index) {
      weights_data[index] = weights_data[index]/total_weight_host;
    },
    num_walkers,
    context);
  
  mgpu::mem_t<float> should_be_one(1, context);
  mgpu::mem_t<float> weights_prefix_sum(num_walkers, context);
  mgpu::scan(weights.data(), num_walkers, weights_prefix_sum.data(),
             mgpu::plus_t<float>(), should_be_one.data(), context);

  float one = mgpu::from_mem(should_be_one)[0];
  printf("%f\n", one);

  mgpu::mem_t<float> probabilities(num_walkers, context);
  auto probabilities_data = probabilities.data();
  mgpu::transform(
    [=]MGPU_DEVICE(uint index) {
      float random = gpu_random::uniforms<1>(uint4{index, 0, 0, 0}, uint2{seed, 1})[0];
      probabilities_data[index] = random;
    }, num_walkers, context);

  mgpu::mergesort(probabilities.data(), num_walkers, mgpu::less_t<float>(), context);

  mgpu::mem_t<int> indices(num_walkers, context);
  mgpu::sorted_search<mgpu::bounds_upper>(probabilities.data(),
                                          num_walkers,
                                          weights_prefix_sum.data(),
                                          num_walkers,
                                          indices.data(),
                                          mgpu::less_t<float>(),
                                          context);

  std::vector<float> ww = mgpu::from_mem(weights);
  std::vector<float> prefixes = mgpu::from_mem(weights_prefix_sum), probs = mgpu::from_mem(probabilities);
  std::vector<int> inds    = mgpu::from_mem(indices);

  for(int ii = 0; ii < num_walkers; ++ii) {
    printf("%f %f %f %d\n", ww[ii], prefixes[ii], probs[ii], inds[ii]);
  }
  
  
  /*
  
  
  auto numbers_data = numbers.data();
  mgpu::transform(
    [=]MGPU_DEVICE(uint index) {
      numbers_data[index] = -float(index % 5);
    }, 20, context);

  std::vector<float> before = mgpu::from_mem(numbers);
  for(auto xx : before)
    printf("%f ", xx);
  printf("\n");

  mgpu::mergesort(numbers.data(), 20, mgpu::less_t<float>(), context);

  std::vector<float> after = mgpu::from_mem(numbers);
  for(auto xx : after)
    printf("%f ", xx);
  printf("\n");
  */  
  return 0;
}
