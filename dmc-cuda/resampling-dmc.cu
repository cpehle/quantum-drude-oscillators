#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <moderngpu/kernel_mergesort.hxx>
#include <moderngpu/kernel_sortedsearch.hxx>
#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_reduce.hxx>
#include <moderngpu/kernel_scan.hxx>

#include "random-numbers.hxx"

#include "harmonic-oscillator.hxx"

using system_t = importance_sampled_harmonic_oscillator<1>;
using walker_state_t = typename system_t::walker_state_t;
using parameter_t = typename system_t::parameter_t;

int main(int argc, char **argv) {
  mgpu::standard_context_t context;
  uint seed = 11;

  assert(argc == 3);
  uint niters = atoi(argv[1]);
  int num_walkers = atoi(argv[2]);

  float dt = 0.001;
  float sqrt_dt = sqrt(dt);

  auto guide_parameters = parameter_t{0.0};  

  mgpu::mem_t<walker_state_t> old_walker_state(num_walkers, context);
  mgpu::mem_t<walker_state_t> new_walker_state(num_walkers, context);

  // Initialize walker positions to random gaussians of std 1
  auto old_walker_state_data = old_walker_state.data();
  mgpu::transform(
    [=]MGPU_DEVICE(uint index) {
      auto randoms =
        gpu_random::uniforms<system_t::walker_dimension>(uint4{index, 0, 0, 0},
                                                         uint2{seed, 0});
      old_walker_state_data[index] = randoms;
    }, num_walkers, context);  
  
  mgpu::mem_t<float> weights(num_walkers, context);

  for (uint local_iter = 0; local_iter < niters; ++local_iter) {
    old_walker_state_data = old_walker_state.data();
    mgpu::mem_t<float2> energy_estimate_device(1, context);
  
    /* Perform diffusion and compute the weight for every walker. */
    auto weights_data = weights.data();
    mgpu::transform_reduce(
      [=]MGPU_DEVICE(uint index) {
        auto walker_state = old_walker_state_data[index];          

        auto diffusion_randoms =
          gpu_random::gaussians<system_t::walker_dimension>(uint4{index, local_iter, 0, 0},
                                                            uint2{seed, 1});
        
        // Energy evaluation: evaluate the Hamiltonian at each walker's
        // position.
        auto energy_before = system_t::local_energy(walker_state, guide_parameters);

        //auto drift = system_t::drift_velocity(walker_state, guide_parameters);
        // Diffusion: add a random gaussian of stddev dt to each walker's position;
        // Drift: add dt*drift velocity from guide wavefunction
        mgpu::iterate<system_t::walker_dimension>([&](uint dimension_index) {
            walker_state[dimension_index] += sqrt_dt * diffusion_randoms.values[dimension_index]
              ;//    + dt * drift[dimension_index];
          });
        auto energy_after = system_t::local_energy(walker_state, guide_parameters);

        old_walker_state_data[index] = walker_state;
        
        float weight = exp(-dt * (0.5*(energy_before + energy_after)));
        weights_data[index] = weight;
        return float2{weight, weight * energy_after};
      },
      num_walkers,
      energy_estimate_device.data(),
      plus_float2_t(),
      context);
    float2 energy_estimate_host = mgpu::from_mem(energy_estimate_device)[0];
    assert(!std::isnan(energy_estimate_host.x));
    assert(std::isfinite(energy_estimate_host.x));
    assert(!std::isnan(energy_estimate_host.y));
    assert(std::isfinite(energy_estimate_host.y));
    assert(energy_estimate_host.x != 0);
    
    float energy_estimate = energy_estimate_host.y / energy_estimate_host.x;
    printf("%f %d\n", energy_estimate, num_walkers);    
    assert(!std::isnan(energy_estimate));
    assert(std::isfinite(energy_estimate));

    float total_weight_host = energy_estimate_host.x;

    /* Compute the prefix-sum for weights. */
    mgpu::mem_t<float> should_be_one(1, context);
    mgpu::mem_t<float> weights_prefix_sum(num_walkers, context);
    mgpu::transform_scan<float,mgpu::scan_type_t::scan_type_inc>(
      [=]MGPU_DEVICE(uint index) {
        return weights_data[index]/total_weight_host;
      },
      num_walkers,
      weights_prefix_sum.data(),
      mgpu::plus_t<float>(),
      should_be_one.data(),
      context);
  
    float one = mgpu::from_mem(should_be_one)[0];
    assert(abs(one-1.0) < 1e-6);

    /*
      as in, if you want to have 4 walkers, then use the points (0.2+x,0.4+x,0.6+x,0.8+x), where x is uniform between 0 and 0.2
    */
//    float jitter = 0;//gpu_random::uniforms<1>(uint4{0, local_iter, 0, 0}, uint2{seed, 2})[0];
    mgpu::mem_t<float> probabilities(num_walkers, context);
    auto probabilities_data = probabilities.data();
    mgpu::transform(
      [=]MGPU_DEVICE(uint index) {
        float random = gpu_random::uniforms<1>(uint4{index, local_iter, 0, 0}, uint2{seed, 2})[0];
        //      float random = float(index+1)/(num_walkers+1) + jitter;
        probabilities_data[index] = random;
      }, num_walkers, context);

    mgpu::mergesort(probabilities.data(), num_walkers, mgpu::less_t<float>(), context);

    auto new_walker_state_data = new_walker_state.data();
    mgpu::sorted_search<mgpu::bounds_lower>(
      probabilities.data(), num_walkers,
      weights_prefix_sum.data(), num_walkers,
      mgpu::make_store_iterator<int>([=]MGPU_DEVICE(uint parent_index, uint child_index) {
          new_walker_state_data[child_index] = old_walker_state_data[parent_index];
        }),
      mgpu::less_t<float>(),
      context);

    old_walker_state.swap(new_walker_state);
  }

  /*
    std::vector<float> ww = mgpu::from_mem(weights);
    std::vector<float> prefixes = mgpu::from_mem(weights_prefix_sum), probs = mgpu::from_mem(probabilities);

    for(int ii = 0; ii < num_walkers; ++ii) {
    printf("%f %f %f\n", ww[ii], prefixes[ii], probs[ii]);
    }
  */
  return 0;
}
