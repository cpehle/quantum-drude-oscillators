#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <moderngpu/kernel_mergesort.hxx>
#include <moderngpu/kernel_sortedsearch.hxx>
#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_reduce.hxx>
#include <moderngpu/kernel_scan.hxx>

#include "random-numbers.hxx"

int main(int argc, char **argv) {
  mgpu::standard_context_t context;
  uint seed = 11;
  uint num_walkers = 20;

  mgpu::mem_t<float> weights(num_walkers, context);
  mgpu::mem_t<float> total_weight(1, context);

  /* Compute the weight for every walker. */
  auto weights_data = weights.data();
  mgpu::transform_reduce(
    [=]MGPU_DEVICE(uint index) {
      float weight = index*index;
      weights_data[index] = weight;
      return weight;
    },
    num_walkers,
    total_weight.data(),
    mgpu::plus_t<float>(),
    context);

  float total_weight_host = mgpu::from_mem(total_weight)[0];

  /* Compute the prefix-sum for weights. */
  mgpu::mem_t<float> should_be_one(1, context);
  mgpu::mem_t<float> weights_prefix_sum(num_walkers, context);
  mgpu::transform_scan<float,mgpu::scan_type_t::scan_type_inc>(
    [=]MGPU_DEVICE(uint index) {
      return weights_data[index]/total_weight_host;
    },
    num_walkers,
    weights_prefix_sum.data(),
    mgpu::plus_t<float>(),
    should_be_one.data(),
    context);
  
  float one = mgpu::from_mem(should_be_one)[0];
  printf("%f\n", one);

  /*
    as in, if you want to have 4 walkers, then use the points (0.2+x,0.4+x,0.6+x,0.8+x), where x is uniform between 0 and 0.2
  */
  mgpu::mem_t<float> probabilities(num_walkers, context);
  auto probabilities_data = probabilities.data();
  mgpu::transform(
    [=]MGPU_DEVICE(uint index) {
      //float random = gpu_random::uniforms<1>(uint4{index, 0, 0, 0}, uint2{seed, 1})[0];
      float random = float(index)/num_walkers;
      probabilities_data[index] = random;
    }, num_walkers, context);

  mgpu::mergesort(probabilities.data(), num_walkers, mgpu::less_t<float>(), context);

  mgpu::mem_t<int> indices(num_walkers, context);
  mgpu::sorted_search<mgpu::bounds_lower>(
    probabilities.data(), num_walkers,
    weights_prefix_sum.data(), num_walkers,
    mgpu::make_store_iterator<int>([=]MGPU_DEVICE(uint parent_index, uint child_index) {
        printf("%d %d\n", child_index, parent_index);
      }),
    mgpu::less_t<float>(),
    context);
  
  std::vector<float> ww = mgpu::from_mem(weights);
  std::vector<float> prefixes = mgpu::from_mem(weights_prefix_sum), probs = mgpu::from_mem(probabilities);
  //std::vector<int> inds    = mgpu::from_mem(indices);

  for(int ii = 0; ii < num_walkers; ++ii) {
    printf("%f %f %f\n", ww[ii], prefixes[ii], probs[ii]);
  }
 
  return 0;
}
