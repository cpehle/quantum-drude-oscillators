#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <moderngpu/kernel_mergesort.hxx>
#include <moderngpu/kernel_sortedsearch.hxx>
#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_reduce.hxx>
#include <moderngpu/kernel_scan.hxx>

#include "random-numbers.hxx"

//#include "harmonic-oscillator.hxx"
//#include "helium.hxx"
#include "qdo-dimer.hxx"

using num_t = double;

//using system_t = importance_sampled_harmonic_oscillator<3>;
//using system_t = helium;
using system_t = qdo_atom_dimer;
using walker_state_t = typename system_t::walker_state_t;
using parameter_t = typename system_t::parameter_t;

int main(int argc, char **argv) {
  mgpu::standard_context_t context;
  uint seed = 11;

  assert(argc == 3);
  uint niters = atoi(argv[1]);
  int num_walkers = atoi(argv[2]);

  double dt = 0.01;
  double sqrt_dt = sqrt(dt);

  auto guide_parameters = parameter_t{2.0};

  mgpu::mem_t<walker_state_t> old_walker_state(num_walkers, context);
  mgpu::mem_t<walker_state_t> new_walker_state(num_walkers, context);

  // Initialize walker positions to random gaussians of std 1
  auto old_walker_state_data = old_walker_state.data();
  mgpu::transform(
    [=]MGPU_DEVICE(uint index) {
      auto randoms =
        gpu_random::gaussians<system_t::walker_dimension>(uint4{index, 0, 0, 0},
                                                         uint2{seed, 0});
      old_walker_state_data[index] = randoms;
    }, num_walkers, context);  
  
  mgpu::mem_t<double> weights(num_walkers, context);
  mgpu::mem_t<double> probabilities(num_walkers, context);
  mgpu::mem_t<double> weights_prefix_sum(num_walkers, context);

  double total_energy = 0.0, total_squared_energy = 0.0;
  
  for (uint local_iter = 0; local_iter < niters; ++local_iter) {
    old_walker_state_data = old_walker_state.data();
    mgpu::mem_t<double2> energy_estimate_device(1, context);
  
    /* Perform diffusion and compute the weight for every walker. */
    auto weights_data = weights.data();
    mgpu::transform_reduce(
      [=]MGPU_DEVICE(uint index) {
        auto walker_state = old_walker_state_data[index];          

        auto diffusion_randoms =
          gpu_random::gaussians<system_t::walker_dimension>(uint4{index, local_iter, 0, 0},
                                                            uint2{seed, 1});
        
        // Energy evaluation: evaluate the Hamiltonian at each walker's
        // position.
        auto energy_before = system_t::local_energy(walker_state, guide_parameters);

        auto drift = system_t::drift_velocity(walker_state, guide_parameters);
        // Diffusion: add a random gaussian of stddev dt to each walker's position;
        // Drift: add dt*drift velocity from guide wavefunction
        mgpu::iterate<system_t::walker_dimension>([&](uint dimension_index) {
            walker_state[dimension_index] += sqrt_dt * diffusion_randoms.values[dimension_index]
              + dt * drift[dimension_index];
          });
        auto energy_after = system_t::local_energy(walker_state, guide_parameters);

        old_walker_state_data[index] = walker_state;
        
        double weight = exp(-dt * (0.5*(energy_before + energy_after)));
        weights_data[index] = weight;
        return double2{weight, weight * energy_after};
      },
      num_walkers,
      energy_estimate_device.data(),
      plus_double2_t(),
      context);
    double2 energy_estimate_host = mgpu::from_mem(energy_estimate_device)[0];
    assert(!std::isnan(energy_estimate_host.x));
    assert(std::isfinite(energy_estimate_host.x));
    assert(!std::isnan(energy_estimate_host.y));
    assert(std::isfinite(energy_estimate_host.y));
    assert(energy_estimate_host.x != 0);
    
    double energy_estimate = energy_estimate_host.y / energy_estimate_host.x;
    //printf("%f %d\n", energy_estimate, num_walkers);    
    assert(!std::isnan(energy_estimate));
    assert(std::isfinite(energy_estimate));

    double total_weight_host = energy_estimate_host.x;

    /* Compute the prefix-sum for weights. */
    mgpu::mem_t<double> should_be_one(1, context);
    mgpu::transform_scan<double,mgpu::scan_type_t::scan_type_inc>(
      [=]MGPU_DEVICE(uint index) {
        return weights_data[index]/total_weight_host;
      },
      num_walkers,
      weights_prefix_sum.data(),
      mgpu::plus_t<double>(),
      should_be_one.data(),
      context);

    //double weight_sum = mgpu::from_mem(weights_prefix_sum)[num_walkers-1];
    //printf("%.20f\n", weight_sum);
  
    double one = mgpu::from_mem(should_be_one)[0];
    assert(abs(one-1.0) < 1e-6);

    auto probabilities_data = probabilities.data();
    mgpu::transform(
      [=]MGPU_DEVICE(uint index) {
        double jitter = gpu_random::uniforms<1>(uint4{index, local_iter, 0, 0}, uint2{seed, 2})[0];
        double random = double(index)/(num_walkers+1) + jitter/num_walkers;
        
        //double random = gpu_random::uniforms<1>(uint4{index, local_iter, 0, 0}, uint2{seed, 2})[0];
        probabilities_data[index] = random;
      }, num_walkers, context);

    //mgpu::mergesort(probabilities.data(), num_walkers, mgpu::less_t<double>(), context);

    auto new_walker_state_data = new_walker_state.data();
    auto weights_prefix_sum_data = weights_prefix_sum.data();
    mgpu::sorted_search<mgpu::bounds_lower>(
      probabilities.data(), num_walkers,
      weights_prefix_sum.data(), num_walkers,
      mgpu::make_store_iterator<int>([=]MGPU_DEVICE(uint parent_index, uint child_index) {
          if(parent_index >= num_walkers) {
            parent_index = num_walkers-1;/*
            printf("%d %d %d %.20f %.20f\n", parent_index, child_index,
                   num_walkers,
                   probabilities_data[child_index],
                   weights_prefix_sum_data[parent_index-1]);*/
          }
          assert(parent_index < num_walkers);

          new_walker_state_data[child_index] = old_walker_state_data[parent_index];
        }),
      mgpu::less_t<double>(),
      context);

    old_walker_state.swap(new_walker_state);

    if(local_iter > 1300) {
      total_energy += energy_estimate;
      total_squared_energy += energy_estimate*energy_estimate;
    }
  }

  niters -= 1300;
  double energy_mean = total_energy/niters;
  double energy_variance = total_squared_energy/niters - energy_mean*energy_mean;
  
  printf("%f %f\n", energy_mean, energy_variance);  
  

  /*
    std::vector<double> ww = mgpu::from_mem(weights);
    std::vector<double> prefixes = mgpu::from_mem(weights_prefix_sum), probs = mgpu::from_mem(probabilities);

    for(int ii = 0; ii < num_walkers; ++ii) {
    printf("%f %f %f\n", ww[ii], prefixes[ii], probs[ii]);
    }
  */
  return 0;
}
