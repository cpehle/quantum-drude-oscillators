#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/kernel_load_balance.hxx>
#include <moderngpu/kernel_reduce.hxx>

#include <hiprand/hiprand_kernel.h>
#include <hiprand_normal.h>

const int dim = 3;

struct alignas(8) walker_state_t {
  float pos[dim];
};

MGPU_DEVICE float harmonic_oscillator_hamiltonian(walker_state_t state) {
  float xx;
  for(int ii = 0; ii < dim; ++ii)
    xx += state.pos[ii]*state.pos[ii];
  return xx/2;
}

int main(int argc, char** argv) {
  assert(argc == 3);
  uint niters = atoi(argv[1]);
  int target_num_walkers = atoi(argv[2]);
  
  uint seed = 10;
  float dt = 0.05;
  float damping_alpha = 0.1;

  float sqrt_dt = sqrt(dt);  
  int num_walkers = target_num_walkers;
  float target_energy = 1.0;

  mgpu::standard_context_t context;
  
  mgpu::mem_t<walker_state_t> old_walker_state(num_walkers, context);
  auto old_walker_state_data = old_walker_state.data();
  
  // Initialize all walker positions to 0
  mgpu::transform(
    [=]MGPU_DEVICE(int index) {
      for(int ii = 0; ii < dim; ++ii)
        old_walker_state_data[index].pos[ii] = 0;
    }, num_walkers, context);
    
  for(uint iter = 0; iter < niters; ++iter) {
    printf("%d %f\n", num_walkers, target_energy);
    old_walker_state_data = old_walker_state.data();

    // Diffusion: add a random gaussian of stddev dt to each walker's position
    mgpu::transform(
      [=]MGPU_DEVICE(uint index) {
        uint4 result = curand_Philox4x32_10(uint4{index, iter, 0, 0}, uint2{seed, 0});
        
        float2 hi = _curand_box_muller(result.x, result.y);
        float2 lo = _curand_box_muller(result.z, result.w);

        old_walker_state_data[index].pos[0] += sqrt_dt * hi.x;
        old_walker_state_data[index].pos[1] += sqrt_dt * hi.y;
        old_walker_state_data[index].pos[2] += sqrt_dt * lo.x;

      }, num_walkers, context);

    // Energy evaluation: evaluate the Hamiltonian at each walker's
    // position
    mgpu::mem_t<float> energy(num_walkers, context);
    auto energy_data = energy.data();
  
    mgpu::transform(
      [=]MGPU_DEVICE(int index) {
        energy_data[index] = harmonic_oscillator_hamiltonian(old_walker_state_data[index]);
      }, num_walkers, context);
    
    // Birth-death I: calculate the number of copies of each walker in the
    // next generation
    mgpu::mem_t<int> children(num_walkers, context);
    int* children_data = children.data();

    mgpu::mem_t<float> energy_estimate(1, context);
    mgpu::transform_reduce(
      [=]MGPU_DEVICE(uint index) {
        float branching_factor = exp(-dt * (energy_data[index] - target_energy));
        uint4 rand_result = curand_Philox4x32_10(uint4{index, iter, 1, 0}, uint2{seed, 0});
        float uniform_float = _curand_uniform(rand_result.x);

        children_data[index] = int(branching_factor + uniform_float);

        return branching_factor * energy_data[index];
      }, num_walkers, energy_estimate.data(), mgpu::plus_t<float>(), context);

    // Birth-death II: compute a prefix-sum of the number-of-copies for
    // each walker
    mgpu::mem_t<int> children_offsets(num_walkers, context);
    mgpu::mem_t<int> total_children(1, context);
    mgpu::scan(children.data(), num_walkers, children_offsets.data(),
               mgpu::plus_t<int>(), total_children.data(), context);

    // Birth-death III: Now create children-number of copies of each
    // walker into a second array.
    int num_children = from_mem(total_children)[0];
    assert(num_children > 0);

    mgpu::mem_t<int> next_gen(num_children, context);
    int* next_gen_data = next_gen.data();

    // Fill next_gen with the value of the parent walker.
    mgpu::mem_t<walker_state_t> new_walker_state(num_children, context);
    auto new_state = new_walker_state.data();

    mgpu::transform_lbs(
      [=]MGPU_DEVICE(int index, int parent, int sibling,
                     mgpu::tuple<walker_state_t> desc) {
        new_state[index] = mgpu::get<0>(desc);
      }, num_children, children_offsets.data(), num_walkers, 
      mgpu::make_tuple(old_walker_state.data()),
      context
      );
  
    old_walker_state.swap(new_walker_state);
    num_walkers = num_children;
    target_energy += damping_alpha * (log(target_num_walkers) - log(num_walkers));
  }  
  return 0;
}

// nvcc -gencode arch=compute_52,code=sm_52 -std=c++11 -I libs/moderngpu/src --expt-extended-lambda -Xptxas="-v" -lineinfo
