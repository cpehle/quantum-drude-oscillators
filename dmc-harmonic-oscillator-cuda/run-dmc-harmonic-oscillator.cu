#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/kernel_load_balance.hxx>

#include <hiprand/hiprand_kernel.h>
#include <hiprand_normal.h>

const int dim = 3;

struct alignas(8) walker_state_t {
  float pos[dim];
};

MGPU_DEVICE float harmonic_oscillator_hamiltonian(walker_state_t state) {
  float xx;
  for(int ii = 0; ii < dim; ++ii)
    xx += state.pos[ii]*state.pos[ii];
  return xx/2;
}

int main(int argc, char** argv) {
  mgpu::standard_context_t context;
  int seed = 10;
  int target_num_walkers = 100;
  float dt = 0.01;

  // for(int iter = 0; iter < 1 million; ++iter)
  
  int num_walkers = target_num_walkers;
  float target_energy = 0.0;

  mgpu::mem_t<walker_state_t> old_walker_state(num_walkers, context);
  auto old_walker_state_data = old_walker_state.data();
  
  // Initialize all walker positions to 0
  mgpu::transform(
    [=]MGPU_DEVICE(int index) {
      for(int ii = 0; ii < dim; ++ii)
        old_walker_state_data[index].pos[ii] = 0;
    }, num_walkers, context);

  // Diffusion: add a random gaussian to each walker's position
  // TODO: times sqrt_t
  mgpu::transform(
    [=]MGPU_DEVICE(int index) {
      hiprandStatePhilox4_32_10_t state;
      
      // ask JohnS if I can just seed++ each iteration, maybe sequence++
      hiprand_init(seed, 0, index, &state); 
      uint4 result = curand4(&state);

      float2 hi = _curand_box_muller(result.x, result.y);
      float2 lo = _curand_box_muller(result.z, result.w);

      old_walker_state_data[index].pos[0] += hi.x;
      old_walker_state_data[index].pos[1] += hi.y;
      old_walker_state_data[index].pos[2] += lo.x;
      // old_walker_state_data[index].pos[3] += lo.y;
    }, num_walkers, context);

  // Energy evaluation: evaluate the Hamiltonian at each walker's
  // position
  mgpu::mem_t<float> energy(num_walkers, context);
  auto energy_data = energy.data();
  
  mgpu::transform(
    [=]MGPU_DEVICE(int index) {
      energy_data[index] = harmonic_oscillator_hamiltonian(old_walker_state_data[index]);
    }, num_walkers, context);

  // Birth-death I: calculate the number of copies of each walker in the
  // next generation
  mgpu::mem_t<int> children(num_walkers, context);
  int* children_data = children.data();

  mgpu::transform(
    [=]MGPU_DEVICE(int index) {
      children_data[index] = exp(-dt * (energy_data[index] - target_energy));
      // TODO plus a uniformly distributed integer in [0,1]
    }, num_walkers, context);

  // Birth-death II: compute a prefix-sum of the number-of-copies for
  // each walker
  mgpu::mem_t<int> children_offsets(num_walkers, context);
  mgpu::mem_t<int> total_children(1, context);
  mgpu::scan(children.data(), num_walkers, children_offsets.data(),
             mgpu::plus_t<int>(), total_children.data(), context);

  // Birth-death III: Now create children-number of copies of each
  // walker into a second array.
  int num_children = from_mem(total_children)[0];

  mgpu::mem_t<int> next_gen(num_children, context);
  int* next_gen_data = next_gen.data();

  // Fill next_gen with the value of the parent walker.
  mgpu::mem_t<walker_state_t> new_walker_state(num_children, context);
  auto new_state = new_walker_state.data();

  mgpu::transform_lbs(
    [=]MGPU_DEVICE(int index, int parent, int sibling,
                   mgpu::tuple<walker_state_t> desc) {
      new_state[index] = mgpu::get<0>(desc);
    }, num_children, children_offsets.data(), num_walkers, 
    mgpu::make_tuple(old_walker_state.data()),
    context
  );
  
  old_walker_state.swap(new_walker_state);
  num_walkers = num_children;
  
  return 0;
}
