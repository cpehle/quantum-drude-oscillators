#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/kernel_load_balance.hxx>
#include <moderngpu/kernel_reduce.hxx>

#include "random-numbers.hxx"

struct plus_float2_t : public std::binary_function<float2, float2, float2> {
  MGPU_HOST_DEVICE float2 operator()(float2 a, float2 b) const {
    return float2{a.x + b.x, a.y + b.y};
  }
};

template<int Dimension_>
struct quantum_system_t {
  enum { Dimension = Dimension_};

  struct alignas(8) walker_state_t {
    float pos[Dimension];
  };
};

template<int Dim>
struct harmonic_oscillator : quantum_system_t<Dim> {
  using walker_state_t = typename quantum_system_t<Dim>::walker_state_t;
  MGPU_DEVICE static float local_energy(walker_state_t state) {
    float xx = 0;
    for(int ii = 0; ii < Dim; ++ii)
      xx += state.pos[ii]*state.pos[ii];
    return xx/2;
  }
};

struct helium : quantum_system_t<6> {
  MGPU_DEVICE static float local_energy(walker_state_t state) {
    /* r1,r2,r12 = norm(pos[0]), norm(pos[1]), norm(pos[0]-pos[1])
       return 1/r12 - 2/r1 - 2/r2
    */

    float r1=0, r2=0, r12=0, tmp;
    for(int ii = 0; ii < 3; ++ii) {
      r1  += state.pos[ii] * state.pos[ii];
      r2  += state.pos[ii+3] * state.pos[ii+3];

      tmp = state.pos[ii+3] - state.pos[ii];
      r12 += tmp*tmp;
    }
    r1 = sqrt(r1); r2 = sqrt(r2); r12 = sqrt(r12);
    return 1/r12 - 2/r1 - 2/r2;
  }
};

template<typename tt>
struct DMC {
  using system_t = tt;
  using walker_state_t = typename system_t::walker_state_t;
  
  mgpu::mem_t<walker_state_t> old_walker_state;
  mgpu::context_t & context;
  uint seed = 10;
  float dt = 0.005;
  float damping_alpha = 0.1;
  float sqrt_dt = sqrt(dt);
  uint num_walkers;
  uint iter = 0;
  float target_energy = 0.0;
  int target_num_walkers;
  
  DMC(int target_num_walkers, mgpu::context_t & context) :
    num_walkers(target_num_walkers),
    target_num_walkers(target_num_walkers),
    context(context),
    old_walker_state(target_num_walkers, context)
  {}

  void initialize() {
    auto old_walker_state_data = old_walker_state.data();
  
    // Initialize all walker positions to random gaussians of std 1
    mgpu::transform(
      [=]MGPU_DEVICE(uint index) {
        auto randoms = gpu_random::uniforms<system_t::Dimension>(uint4{index, 0, 0, 0}, uint2{seed, 0});
        mgpu::iterate<system_t::Dimension>([&](int dimension_index) {
            old_walker_state_data[index].pos[dimension_index] = randoms[dimension_index];
          });
      }, num_walkers, context);
  }

  float step() {
    auto old_walker_state_data = old_walker_state.data();
    mgpu::mem_t<int> children(num_walkers, context);
    int* children_data = children.data();
    
    mgpu::mem_t<float2> energy_estimate_device(1, context);
    
    mgpu::transform_reduce(
      [=]MGPU_DEVICE(uint index) {
        auto walker_state = old_walker_state_data[index];          

        auto diffusion_randoms = gpu_random::gaussians<system_t::Dimension>(uint4{index, iter, 0, 0}, uint2{seed, 1});
        
        // Energy evaluation: evaluate the Hamiltonian at each walker's
        // position.
        auto energy_before = system_t::local_energy(walker_state);

        // Diffusion: add a random gaussian of stddev dt to each walker's position
        mgpu::iterate<system_t::Dimension>([&](uint dimension_index) {
            walker_state.pos[dimension_index] += sqrt_dt * diffusion_randoms.values[dimension_index];
          });
        auto energy_after = 0; //helium_hamiltonian(walker_state);

        old_walker_state_data[index] = walker_state;
        // Birth-death I: calculate the number of copies of each walker in the
        // next generation

        // We will also estimate the average energy at this point since it
        // uses the branching-factor used to compute birth and death.
        float branching_factor = exp(-dt * (0.5*(energy_before + energy_after) - target_energy));

        auto branching_random = gpu_random::uniforms<1>(uint4{index, iter, 0, 0}, uint2{seed, 2})[0];

        children_data[index] = int(branching_factor + branching_random);

        return float2{branching_factor, branching_factor * energy_after};        
        
      }, num_walkers,
      energy_estimate_device.data(),
      plus_float2_t(),
      context);

    float2 energy_estimate_host = mgpu::from_mem(energy_estimate_device)[0];
    assert(!std::isnan(energy_estimate_host.x));
    assert(std::isfinite(energy_estimate_host.x));
    assert(!std::isnan(energy_estimate_host.y));
    assert(std::isfinite(energy_estimate_host.y));
    assert(energy_estimate_host.x != 0);
    
    float energy_estimate = energy_estimate_host.y / energy_estimate_host.x;
    printf("%f %d %f\n", energy_estimate, num_walkers, target_energy);    
    assert(!std::isnan(energy_estimate));
    assert(std::isfinite(energy_estimate));
    

    // Birth-death II: compute a prefix-sum of the number-of-copies for
    // each walker
    mgpu::mem_t<int> children_offsets(num_walkers, context);
    mgpu::mem_t<int> total_children(1, context);
    mgpu::scan(children.data(), num_walkers, children_offsets.data(),
               mgpu::plus_t<int>(), total_children.data(), context);

    // Birth-death III: Now create children-number of copies of each
    // walker into a second array.
    int num_children = mgpu::from_mem(total_children)[0];
    assert(num_children > 0);

    // Fill new_walker_state with the value of the parent walker.
    mgpu::mem_t<walker_state_t> new_walker_state(num_children, context);
    auto new_state = new_walker_state.data();

    mgpu::transform_lbs(
      [=]MGPU_DEVICE(int index, int parent, int sibling,
                     mgpu::tuple<walker_state_t> desc) {
        new_state[index] = mgpu::get<0>(desc);
      }, num_children, children_offsets.data(), num_walkers, 
      mgpu::make_tuple(old_walker_state.data()),
      context
      );

    old_walker_state.swap(new_walker_state);
    num_walkers = num_children;
    
    if(iter % 100 == 0)
      target_energy = energy_estimate;
    else
      target_energy += damping_alpha * (log(target_num_walkers) - log(num_walkers));

    iter++;
    return energy_estimate;
  }
};

int main(int argc, char** argv) {
  assert(argc == 3);
  uint niters = atoi(argv[1]);
  int target_num_walkers = atoi(argv[2]);
  mgpu::standard_context_t context;
  
  DMC<harmonic_oscillator<3>> dd(target_num_walkers, context);
  dd.initialize();
    
  for(uint iter = 0; iter < niters; ++iter) {
    dd.step();
  }  
  return 0;
}

// nvcc -gencode arch=compute_52,code=sm_52 -std=c++11 -I libs/moderngpu/src --expt-extended-lambda -Xptxas="-v" -lineinfo
