#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_scan.hxx>
#include <moderngpu/kernel_load_balance.hxx>
#include <moderngpu/kernel_reduce.hxx>

#include <hiprand/hiprand_kernel.h>
#include <hiprand_normal.h>

const int walker_dimension = 3;

struct alignas(8) walker_state_t {
  float pos[walker_dimension];
};

MGPU_DEVICE float harmonic_oscillator_hamiltonian(walker_state_t state) {
  float xx;
  for(int ii = 0; ii < walker_dimension; ++ii)
    xx += state.pos[ii]*state.pos[ii];
  return xx/2;
}

struct plus_float2_t : public std::binary_function<float2, float2, float2> {
  MGPU_HOST_DEVICE float2 operator()(float2 a, float2 b) const {
    return float2{a.x + b.x, a.y + b.y};
  }
};

struct array_float_4_t {
  float values[4];
};

MGPU_DEVICE array_float_4_t random_gaussians(uint4 counter, uint2 key) {
  uint4 result = curand_Philox4x32_10(counter, key);
  float2 hi = _curand_box_muller(result.x, result.y);
  float2 lo = _curand_box_muller(result.z, result.w);
  array_float_4_t answer;
  answer.values[0] = hi.x; answer.values[1] = hi.y; answer.values[2] = lo.x; answer.values[3] = lo.y;
  return answer;
}

int main(int argc, char** argv) {
  assert(argc == 3);
  uint niters = atoi(argv[1]);
  int target_num_walkers = atoi(argv[2]);
  
  uint seed = 10;
  float dt = 0.05;
  float damping_alpha = 0.1;

  float sqrt_dt = sqrt(dt);  
  int num_walkers = target_num_walkers;
  float target_energy = 1.0;

  mgpu::standard_context_t context;
  
  mgpu::mem_t<walker_state_t> old_walker_state(num_walkers, context);
  auto old_walker_state_data = old_walker_state.data();
  
  // Initialize all walker positions to random gaussians of std 1
  mgpu::transform(
    [=]MGPU_DEVICE(uint index) {
      auto randoms = random_gaussians(uint4{index, 0, 0, 0}, uint2{seed, 1});
      mgpu::iterate<walker_dimension>([&](int dimension_index) {
          old_walker_state_data[index].pos[dimension_index] = randoms.values[dimension_index];
        });
    }, num_walkers, context);
    
  for(uint iter = 0; iter < niters; ++iter) {
    old_walker_state_data = old_walker_state.data();
    mgpu::mem_t<int> children(num_walkers, context);
    int* children_data = children.data();
    
    mgpu::mem_t<float2> energy_estimate_device(1, context);
    
    mgpu::transform_reduce(
      [=]MGPU_DEVICE(uint index) {
        auto walker_state = old_walker_state_data[index];          
        
        auto randoms = random_gaussians(uint4{index, iter, 0, 0}, uint2{seed, 0});
        
        // Energy evaluation: evaluate the Hamiltonian at each walker's
        // position.
        auto energy_before = harmonic_oscillator_hamiltonian(walker_state);

        // Diffusion: add a random gaussian of stddev dt to each walker's position
        mgpu::iterate<walker_dimension>([&](int dimension_index) {
            walker_state.pos[dimension_index] += sqrt_dt * randoms.values[dimension_index];
          });
        auto energy_after = harmonic_oscillator_hamiltonian(walker_state);

        old_walker_state_data[index] = walker_state;
        // Birth-death I: calculate the number of copies of each walker in the
        // next generation

        // We will also estimate the average energy at this point since it
        // uses the branching-factor used to compute birth and death.
        float branching_factor = exp(-dt * (0.5*(energy_before + energy_after) - target_energy));
        uint4 rand_result = curand_Philox4x32_10(uint4{index, iter, 1, 0}, uint2{seed, 0});
        float uniform_float = _curand_uniform(rand_result.x);

        children_data[index] = int(branching_factor + uniform_float);

        return float2{branching_factor, branching_factor * energy_after};        
        
      }, num_walkers,
      energy_estimate_device.data(),
      plus_float2_t(),
      context);

    float2 energy_estimate_host = mgpu::from_mem(energy_estimate_device)[0];
    float energy_estimate = energy_estimate_host.y / energy_estimate_host.x;
    printf("%f %d %f\n", energy_estimate, num_walkers, target_energy);

    // Birth-death II: compute a prefix-sum of the number-of-copies for
    // each walker
    mgpu::mem_t<int> children_offsets(num_walkers, context);
    mgpu::mem_t<int> total_children(1, context);
    mgpu::scan(children.data(), num_walkers, children_offsets.data(),
               mgpu::plus_t<int>(), total_children.data(), context);

    // Birth-death III: Now create children-number of copies of each
    // walker into a second array.
    int num_children = mgpu::from_mem(total_children)[0];
    assert(num_children > 0);

    // Fill new_walker_state with the value of the parent walker.
    mgpu::mem_t<walker_state_t> new_walker_state(num_children, context);
    auto new_state = new_walker_state.data();

    mgpu::transform_lbs(
      [=]MGPU_DEVICE(int index, int parent, int sibling,
                     mgpu::tuple<walker_state_t> desc) {
        new_state[index] = mgpu::get<0>(desc);
      }, num_children, children_offsets.data(), num_walkers, 
      mgpu::make_tuple(old_walker_state.data()),
      context
      );
  
    old_walker_state.swap(new_walker_state);
    num_walkers = num_children;
    target_energy += damping_alpha * (log(target_num_walkers) - log(num_walkers));
  }  
  return 0;
}

// nvcc -gencode arch=compute_52,code=sm_52 -std=c++11 -I libs/moderngpu/src --expt-extended-lambda -Xptxas="-v" -lineinfo
