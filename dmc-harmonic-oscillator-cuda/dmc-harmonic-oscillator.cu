#include "hip/hip_runtime.h"
// -*- c++ -*-
#include <moderngpu/transform.hxx>
#include <hiprand/hiprand_kernel.h>

#include "random-numbers.hxx"

template<int Dimension_>
struct quantum_system_t {
  enum { Dimension = Dimension_};

  struct alignas(8) walker_state_t {
    float pos[Dimension];
  };
};

template<int Dim>
struct harmonic_oscillator : quantum_system_t<Dim> {
  using walker_state_t = typename quantum_system_t<Dim>::walker_state_t;
  MGPU_DEVICE static float local_energy(walker_state_t state) {
      return 0;
  }
};

template<typename tt>
struct DMC {
  using system_t = tt;
  using walker_state_t = typename system_t::walker_state_t;
  
  uint seed = 10;
  float dt = 0.005;
  float damping_alpha = 0.1;
  float sqrt_dt = sqrt(dt);
  uint num_walkers;
  uint iter = 0;
  float target_energy = 0.0;
  int target_num_walkers;
  
  DMC(int target) :
    num_walkers(target),
    target_num_walkers(target)
  {}

  void initialize() {
    printf("%d %d\n", system_t::Dimension, num_walkers);
    mgpu::standard_context_t context;
    // Initialize all walker positions to random gaussians of std 1
    mgpu::transform(
      [=]MGPU_DEVICE(uint index) {
          //auto randoms = gpu_random::uniforms<12>(uint4{index, 0, 0, 0}, uint2{seed, 0});
          uint4 result = curand_Philox4x32_10(uint4{index, 0, 0, 0}, uint2{seed, 0});
          //uint4 result{index,seed,3,4};
          printf("HIII %d\n", result.x);
      }, num_walkers, context);

    context.synchronize();
  }

  float step() {
  }
};

int main(int argc, char** argv) {
  assert(argc == 3);
  uint niters = atoi(argv[1]);
  int target_num_walkers = atoi(argv[2]);
  
  DMC<harmonic_oscillator<3>> dd(target_num_walkers);
  dd.initialize();
    
  for(uint iter = 0; iter < niters; ++iter) {
    dd.step();
  }  
  return 0;
}

// nvcc -gencode arch=compute_52,code=sm_52 -std=c++11 -I libs/moderngpu/src --expt-extended-lambda -Xptxas="-v" -lineinfo
