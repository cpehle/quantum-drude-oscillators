#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand_normal.h>
#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_reduce.hxx>
#include <moderngpu/memory.hxx>
#include <cstdlib>

int main(int argc, char** argv) {

  mgpu::standard_context_t context;

  int count = 100000000;
  int seed = atoi(argv[1]);

  mgpu::mem_t<float2> outputs(1, context);

  mgpu::transform_reduce(
    [=]MGPU_DEVICE(int index) {
      hiprandStatePhilox4_32_10_t state;

      hiprand_init(seed, 0, index, &state);
      uint4 result = curand4(&state);

      float2 yo = _curand_box_muller(result.x, result.y);
      return yo;

    }, count, outputs.data(), 
      []MGPU_DEVICE(float2 a, float2 b) {
        return make_float2(a.x + b.x, a.y + b.y);
      }, context
  );

  std::vector<float2> host = from_mem(outputs);
  for(float2 p : host) {
    printf("% 13.3e % 13.3e\n", p.x / count, p.y / count);
  }

  return 0;
}

// nvcc -arch sm_52 -std=c++11 -I libs/moderngpu/src --expt-extended-lambda -Xptxas="-v" -o curand_test curand_test.cu
