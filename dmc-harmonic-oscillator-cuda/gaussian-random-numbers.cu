#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand_normal.h>

#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_reduce.hxx>
#include <moderngpu/memory.hxx>

#include <cstdlib>
#include <cassert>

int main(int argc, char** argv) {
  assert(argc == 2);
  mgpu::standard_context_t context;

  uint count = 10000;
  uint seed = atoi(argv[1]);

  mgpu::mem_t<float2> outputs(count, context);
  auto outputs_data = outputs.data();

  mgpu::transform(
    [=]MGPU_DEVICE(uint index) {
      uint4 c {index, 0, 0, 0};
      uint2 k = {seed, 0};

      uint4 result = curand_Philox4x32_10(c, k);

      float2 yo = _curand_box_muller(result.x, result.y);
      outputs_data[index] = yo;
    },
    count,
    context
  );

  std::vector<float2> host = from_mem(outputs);
  for(float2 p : host) {
    printf("% 13.3e % 13.3e\n", p.x, p.y);
  }

  return 0;
}

// nvcc -arch sm_52 -std=c++11 -I libs/moderngpu/src --expt-extended-lambda -Xptxas="-v" -o curand_test curand_test.cu
