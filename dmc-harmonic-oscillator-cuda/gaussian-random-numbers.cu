#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand_normal.h>

#include <moderngpu/transform.hxx>
#include <moderngpu/kernel_reduce.hxx>
#include <moderngpu/memory.hxx>

#include <cstdlib>
#include <cassert>

int main(int argc, char** argv) {
  assert(argc == 2);
  mgpu::standard_context_t context;

  int count = 1000;
  int seed = atoi(argv[1]);

  mgpu::mem_t<float2> outputs(count, context);
  auto outputs_data = outputs.data();

  mgpu::transform(
    [=]MGPU_DEVICE(int index) {
      hiprandStatePhilox4_32_10_t state;

      hiprand_init(seed, 0, index, &state);
      uint4 result = curand4(&state);

      float2 yo = _curand_box_muller(result.x, result.y);
      outputs_data[index] = yo;
    },
    count,
    context
  );

  std::vector<float2> host = from_mem(outputs);
  for(float2 p : host) {
    printf("% 13.3e % 13.3e\n", p.x, p.y);
  }

  return 0;
}

// nvcc -arch sm_52 -std=c++11 -I libs/moderngpu/src --expt-extended-lambda -Xptxas="-v" -o curand_test curand_test.cu
